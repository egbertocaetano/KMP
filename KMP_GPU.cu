#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <string>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "App.h"

#define TRUE 1
#define FALSE 0

#define CHECK_ERROR(call) do {                                                    \
   if( hipSuccess != call) {                                                             \
      std::cerr << std::endl << "CUDA ERRO: " <<                             \
         hipGetErrorString(call) <<  " in file: " << __FILE__                \
         << " in line: " << __LINE__ << std::endl;                               \
         exit(0);                                                                                 \
   } } while (0)


void partial_match_table(int * pmt, char * word, int len_word)
{
	int pos = 2;
	int cnd = 0;

	pmt[0] = -1;
	pmt[1] = 0;

	while(pos < len_word)
	{
		if(word[pos - 1] == word[cnd])
		{
			cnd = cnd + 1;
			pmt[pos] = cnd;
			pos++; 
		}
		else if(cnd > 0)
		{
			cnd = pmt[cnd];
		}
		else
		{
			pmt[pos] = 0;
			pos++;
		}
	}
}

//__device__ int get_start(int thread, int offset){
//  return thread*offset; 
//}

__device__ int get_end(int thread, int offset){
    return thread*offset+offset;
}

__device__ void KMP_search(int cursor, int end, int len_word, char* word, char* text,int* match, int* pmt){
   
	 //Verify if the distance is bigger that word's length		
	if((end-cursor) >= len_word)
	{
        
		int i = 0;
	    
		while (cursor < end)
		{
	        while(i > -1 && word[i] != text[cursor])
			{
	            i = pmt[i];
	        }

	        cursor++;
	        i++;
	        
			if(i == len_word)
			{
	            match[cursor-i] = 1;
	            i = 0;
	        }
	    }
    }
}

__global__ void kernel_KMP (char* text, char* word, int* pmt, int* match,int len_text, int len_word, 
                            int num_threads,int debug){

	int indice = blockIdx.x * blockDim.x + threadIdx.x;
	int offset = len_text/num_threads;;
	int cursor,end;
	
	if( indice < num_threads)
	{ 
	    cursor = indice*offset;
	    end = indice*offset+offset;
	}
	else
	{ //Threads Secundárias
	    cursor = get_end(indice%num_threads,offset)-(len_word-1);
	    end = get_end(indice%num_threads,offset)+len_word-1;
	    offset = end-cursor;
	}
	
    KMP_search(cursor,end,len_word,word,text,match,pmt);
}

int main(int argc, char **argv)
{
	
	char * word = argv[1];
    int len_word = strlen(word);
    int * pmt = (int*)malloc(len_word*sizeof(int));	
    	
	FILE * file = fopen(argv[2],"r");
    fseek(file,0,SEEK_END);
    int len_text = ftell(file);
    fseek(file,0,0);
    char * text = (char*)malloc(len_text*sizeof(char));
    
	int count = 0;
    
	while(!feof(file)){
        fscanf(file,"%c",&text[count]);
        count++; 
    }



	Stopwatch sw;
    srand(time(NULL));

    int debug = TRUE;
    
  	partial_match_table(pmt, word, len_word);  

	
    int* match = (int*)malloc(len_text*sizeof(int));
    int* d_pmt = NULL;
    int* d_match = NULL;
    char* d_text = NULL;
    char* d_word = NULL;
    int threads = (len_text/len_word);
    int blocks = (threads/1024)+1;
    int threads_per_block = (threads % blocks == 0) ? threads/blocks : (threads/blocks)+1;
  
	CHECK_ERROR(hipMalloc((void**) &d_text, len_text * sizeof(char)));
	CHECK_ERROR(hipMalloc((void**) &d_word, len_word * sizeof(char)));
	CHECK_ERROR(hipMalloc((void**) &d_match, len_text * sizeof(int)));
	CHECK_ERROR(hipMalloc((void**) &d_pmt, len_word * sizeof(int)));
	CHECK_ERROR(hipMemcpy(d_text, text,  len_text * sizeof(char), hipMemcpyHostToDevice));
	CHECK_ERROR(hipMemcpy(d_word, word,  len_word * sizeof(char), hipMemcpyHostToDevice));
    CHECK_ERROR(hipMemcpy(d_pmt, pmt,  len_word * sizeof(int), hipMemcpyHostToDevice));
    CHECK_ERROR(hipMemcpy(d_match, match,  len_text * sizeof(int), hipMemcpyHostToDevice));  

	FREQUENCY(sw);
    START_STOPWATCH(sw);
	
	kernel_KMP<<<blocks, threads_per_block>>> (d_text,d_word,d_pmt,d_match,len_text, len_word, threads/2, debug);
	
	CHECK_ERROR(hipDeviceSynchronize());
	
	STOP_STOPWATCH(sw);
	
	CHECK_ERROR(hipMemcpy(match, d_match, len_text * sizeof(int), hipMemcpyDeviceToHost));
    
    printf("Tamanho do texto: \n%d\n",len_text);
    puts("Numero de ocorrências: ");
    int counter = 0;
	for (int i = 0; i < len_text; i++){
	    if( match[i]){
			counter++;
		    /*printf("%d ",i); 
		    if(counter % 8 == 0)
		        printf("\n");*/
	    }
	}
	printf("\nNúmero de Ocorrências: %d\n",counter);
	printf("Tempo total: %lf ms\n",sw.mElapsedTime);


	//Desalocando as memórias
	CHECK_ERROR(hipFree(d_text));
	CHECK_ERROR(hipFree(d_word));
	CHECK_ERROR(hipFree(d_pmt));
	CHECK_ERROR(hipFree(d_match));
	
	return EXIT_SUCCESS;
};
